#include "ax/core/excepts.hpp"
#include "ax/math/block_matrix/details/cusparse_context.cuh"
#include "csr_compress_impl.hpp"
#include <hipsparse.h>

namespace ax::math::details {

struct Descr {
  hipsparseSpMatDescr_t descr_{nullptr};
  hipsparseDnVecDescr_t rhs_descr_{nullptr};
  hipsparseDnVecDescr_t lhs_descr_{nullptr};
  hipsparseDnVecDescr_t rhs_dst_descr_{nullptr};
  hipsparseDnVecDescr_t lhs_dst_descr_{nullptr};

  Real *buffer_{nullptr};
  size_t buffer_size_{0};

  Descr(size_t rows, size_t cols, size_t nnz, int *row_ptrs, int *col_indices,
        Real *values) {
    auto status =
        hipsparseCreateCsr(&descr_, rows, cols, nnz, row_ptrs, col_indices,
                          values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    if (status != HIPSPARSE_STATUS_SUCCESS) {
      throw make_runtime_error("Failed to create CSR matrix descriptor {}: {}", 
        hipsparseGetErrorName(status), hipsparseGetErrorString(status));
    }

    status = hipsparseCreateDnVec(&rhs_descr_, cols, nullptr, HIP_R_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      throw make_runtime_error("Failed to create dense vector descriptor {}: {}", 
        hipsparseGetErrorName(status), hipsparseGetErrorString(status));
    }

    status = hipsparseCreateDnVec(&lhs_descr_, rows, nullptr, HIP_R_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      throw make_runtime_error("Failed to create dense vector descriptor {}: {}", 
        hipsparseGetErrorName(status), hipsparseGetErrorString(status));
    }

    status = hipsparseCreateDnVec(&rhs_dst_descr_, cols, nullptr, HIP_R_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      throw make_runtime_error("Failed to create dense vector descriptor {}: {}", 
        hipsparseGetErrorName(status), hipsparseGetErrorString(status));
    }

    status = hipsparseCreateDnVec(&lhs_dst_descr_, rows, nullptr, HIP_R_64F);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      throw make_runtime_error("Failed to create dense vector descriptor {}: {}", 
        hipsparseGetErrorName(status), hipsparseGetErrorString(status));
    }
  }

  ~Descr() {
    hipsparseDestroySpMat(descr_);
    hipsparseDestroyDnVec(rhs_descr_);
    hipsparseDestroyDnVec(lhs_descr_);

    if (buffer_) {
      hipFree(buffer_);
      buffer_ = nullptr;
    }
  }
};

std::shared_ptr<void> create_csr_compress_desc_gpu(BufferView<int> row_ptrs,
                                                   BufferView<int> col_indices,
                                                   BufferView<Real> values,
                                                   size_t rows, size_t cols) {
  return std::make_shared<Descr>(rows, cols, values.Shape().X(),
                                 row_ptrs.Data(), col_indices.Data(),
                                 values.Data());
}

void compute_csr_spmv_gpu(BufferView<const Real> x, BufferView<Real> y, Real alpha,
                          Real beta, std::shared_ptr<void> desc) {
  auto descr = std::static_pointer_cast<Descr>(desc);

  if (!is_1d(x.Shape())) {
    throw make_runtime_error("Input vector must be 1D for now.");
  }

  auto status = hipsparseDnVecSetValues(descr->rhs_descr_, const_cast<Real*>(x.Data()));
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("Failed to set rhs dense vector values {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }

  status = hipsparseDnVecSetValues(descr->rhs_dst_descr_, y.Data());
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("Failed to set dst dense vector values {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }

  auto handle = get_cusparse_handle();
  size_t required_buffer_size = 0;
  status = hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, descr->descr_,
      descr->rhs_descr_, &beta, descr->lhs_descr_, HIP_R_64F,
      HIPSPARSE_SPMV_ALG_DEFAULT, &required_buffer_size);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("Failed to get buffer size {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }

  if (descr->buffer_ != nullptr) {
    if (descr->buffer_size_ < required_buffer_size) {
      hipFree(descr->buffer_);
      descr->buffer_ = nullptr;
    }
  }

  if (descr->buffer_ == nullptr) {
    hipMalloc(&descr->buffer_, required_buffer_size);
    descr->buffer_size_ = required_buffer_size;
  }

  status = hipsparseSpMV_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   &alpha, descr->descr_, descr->rhs_descr_,
                                   &beta, descr->rhs_dst_descr_, HIP_R_64F,
                                   HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("Failed to preprocess SPMV {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }

  status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                        descr->descr_, descr->rhs_descr_, &beta,
                        descr->rhs_dst_descr_, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("Failed to compute SPMV {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }
}

} // namespace ax::math::details