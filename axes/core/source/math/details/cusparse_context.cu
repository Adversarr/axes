#include "ax/core/entt.hpp"
#include "ax/core/logging.hpp"
#include "ax/math/details/cusparse_context.cuh"

namespace ax::math::details {

cusparseContext *get_cusparse_handle() {
  return ensure_resource<CuSparseHandle>().handle_;
}

CuSparseHandle::CuSparseHandle() {
  auto status = hipsparseCreate(&handle_);
  AX_CHECK(status == HIPSPARSE_STATUS_SUCCESS,
           "Failed to create cusparse handle, {}: {}",
           hipsparseGetErrorName(status), hipsparseGetErrorString(status));
}

CuSparseHandle::~CuSparseHandle() {
  if (handle_) {
    hipsparseDestroy(handle_);
  }
  handle_ = nullptr;
}

} // namespace ax::math::details