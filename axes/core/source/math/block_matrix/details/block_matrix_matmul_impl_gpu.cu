#include "ax/core/excepts.hpp"
#include "ax/math/block_matrix/details/matmul_impl.hpp"
#include "ax/math/details/cusparse_context.cuh"
#include <hipsparse.h>

namespace ax::math::details {

void block_matrix_matmul_gpu(size_t rows, size_t cols,
                             BufferView<const Real> block_values,
                             BufferView<const int> block_row_ptrs,
                             BufferView<const int> block_col_indices,
                             BufferView<const Real> rhs, BufferView<Real> dst,
                             Real alpha, Real beta, void *descr_type_erased) {
  cusparseContext *handle = get_cusparse_handle();

  size_t nnzb = block_values.Shape().Z();
  size_t block_size = block_values.Shape().X();
  hipsparseMatDescr_t descr = static_cast<hipsparseMatDescr_t>(descr_type_erased);

  auto status = hipsparseDbsrmv(
      handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, rows,
      cols, nnzb, &alpha, descr, block_values.Data(), block_row_ptrs.Data(),
      block_col_indices.Data(), block_size, rhs.Data(), &beta, dst.Data());

  if (status != HIPSPARSE_STATUS_SUCCESS) {
    throw make_runtime_error("hipsparseDbsrmv failed {}: {}",
                             hipsparseGetErrorName(status),
                             hipsparseGetErrorString(status));
  }
}

std::shared_ptr<void> create_bsr_mat_desc_default() {
  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  return std::shared_ptr<void>(descr, [](void *descr) {
    hipsparseDestroyMatDescr(static_cast<hipsparseMatDescr_t>(descr));
  });
}

} // namespace ax::math::details