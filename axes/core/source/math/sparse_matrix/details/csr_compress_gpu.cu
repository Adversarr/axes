#include "ax/core/excepts.hpp"
#include "ax/math/details/cusparse_context.cuh"
#include "csr_compress_impl.hpp"
#include <hipsparse.h>
#include "descriptors.cuh"

namespace ax::math::details {

std::shared_ptr<void> create_csr_compress_desc_gpu(IntBufferView row_ptrs,
                                                   IntBufferView col_indices,
                                                   BufferView<Real> values,
                                                   size_t rows, size_t cols) {
  return std::make_shared<CsrDescr>(rows, cols, values.Shape().X(),
                                 row_ptrs.Data(), col_indices.Data(),
                                 values.Data());
}

void compute_csr_spmv_gpu(BufferView<const Real> x, BufferView<Real> y,
                          Real alpha, Real beta, std::shared_ptr<void> desc) {
  auto descr = std::static_pointer_cast<CsrDescr>(desc);

  if (!is_1d(x.Shape())) {
    AX_THROW_RUNTIME_ERROR("Input vector must be 1D for now.");
  }

  auto status =
      hipsparseDnVecSetValues(descr->rhs_descr_, const_cast<Real *>(x.Data()));
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to set rhs dense vector values {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  status = hipsparseDnVecSetValues(descr->rhs_dst_descr_, y.Data());
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to set dst dense vector values {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  auto handle = get_cusparse_handle();
  size_t required_buffer_size = 0;
  status = hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, descr->sp_descr_,
      descr->rhs_descr_, &beta, descr->rhs_dst_descr_, HIP_R_64F,
      HIPSPARSE_SPMV_ALG_DEFAULT, &required_buffer_size);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to get buffer size {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  if (descr->buffer_ != nullptr) {
    if (descr->buffer_size_ < required_buffer_size) {
      hipFree(descr->buffer_);
      descr->buffer_ = nullptr;
    }
  }

  if (descr->buffer_ == nullptr) {
    hipMalloc(&descr->buffer_, required_buffer_size);
    descr->buffer_size_ = required_buffer_size;
  }

  status = hipsparseSpMV_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   &alpha, descr->sp_descr_, descr->rhs_descr_,
                                   &beta, descr->rhs_dst_descr_, HIP_R_64F,
                                   HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to preprocess SPMV {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                        descr->sp_descr_, descr->rhs_descr_, &beta,
                        descr->rhs_dst_descr_, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to compute SPMV {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }
}

void compute_csr_spmv_transpose_gpu(BufferView<const Real> x,
                                    BufferView<Real> y, Real alpha, Real beta,
                                    std::shared_ptr<void> desc) {
  auto descr = std::static_pointer_cast<CsrDescr>(desc);
  if (!is_1d(x.Shape())) {
    AX_THROW_RUNTIME_ERROR("Input vector must be 1D for now.");
  }

  auto status =
      hipsparseDnVecSetValues(descr->lhs_descr_, const_cast<Real *>(x.Data()));
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to set lhs dense vector values {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  status = hipsparseDnVecSetValues(descr->lhs_dst_descr_, y.Data());
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to set dst dense vector values {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  auto handle = get_cusparse_handle();
  size_t required_buffer_size = 0;
  status = hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, descr->sp_descr_,
      descr->lhs_descr_, &beta, descr->lhs_dst_descr_, HIP_R_64F,
      HIPSPARSE_SPMV_ALG_DEFAULT, &required_buffer_size);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to get buffer size {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  if (descr->buffer_ != nullptr) {
    if (descr->buffer_size_ < required_buffer_size) {
      hipFree(descr->buffer_);
      descr->buffer_ = nullptr;
    }
  }

  if (descr->buffer_ == nullptr) {
    hipMalloc(&descr->buffer_, required_buffer_size);
    descr->buffer_size_ = required_buffer_size;
  }

  status = hipsparseSpMV_preprocess(handle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha,
                                   descr->sp_descr_, descr->lhs_descr_, &beta,
                                   descr->lhs_dst_descr_, HIP_R_64F,
                                   HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to preprocess SPMV {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }

  status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha,
                        descr->sp_descr_, descr->lhs_descr_, &beta,
                        descr->lhs_dst_descr_, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, descr->buffer_);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    AX_THROW_RUNTIME_ERROR("Failed to compute SPMV {}: {}",
                           hipsparseGetErrorName(status),
                           hipsparseGetErrorString(status));
  }
}

} // namespace ax::math::details