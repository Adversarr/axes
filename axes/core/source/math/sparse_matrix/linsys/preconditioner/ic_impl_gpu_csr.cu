#include "../../details/descriptors.cuh"
#include "ax/core/buffer/create_buffer.hpp"
#include "ax/math/buffer_blas.hpp"
#include "ic_impl.hpp"

namespace ax::math {

using namespace details;

ImplIcCsrGpu::ImplIcCsrGpu(ConstRealSparseMatrixPtr mat) : mat_(mat) {}

ImplIcCsrGpu::~ImplIcCsrGpu() = default;

constexpr hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
constexpr hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
constexpr hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
constexpr hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
constexpr hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;
constexpr hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
const double alpha = 1.;
const double one = 1.;
constexpr hipsparseIndexBase_t baseIdx = HIPSPARSE_INDEX_BASE_ZERO;

// refer to the CUDA-samples:
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/cg/cg_example.c

void ImplIcCsrGpu::AnalyzePattern() {
  L_ = mat_->ToCSR();
  L_->Finish();

  size_t rows = mat_->Rows();
  mid_result_ = ensure_buffer<Real>(mid_result_, BufferDevice::Device, {rows});
  temp_descr_ = std::make_shared<DnVec>(rows);
  b_descr_ = std::make_shared<DnVec>(rows);
  x_descr_ = std::make_shared<DnVec>(rows);

  spsv_descr_ = std::make_shared<SpsvDescr>();
  spsv_t_descr_ = std::make_shared<SpsvDescr>();
}

void ImplIcCsrGpu::Factorize() {
  hipsparseMatDescr_t descrM;
  csric02Info_t infoM = NULL;
  int bufferSizeIC = 0;
  void *d_bufferIC;

  auto *cusparseHandle = details::get_cusparse_handle();

  int m = static_cast<size_t>(mat_->Rows());
  int nnz = static_cast<size_t>(prod(mat_->Values()->Shape()));
  // initialize L with the input matrix, because the IC preconditioner
  // has same pattern as the input matrix

  Real *d_L_values = L_->Values()->Data();
  int *d_A_rows = L_->RowPtrs()->Data();
  int *d_A_columns = L_->ColIndices()->Data();

  auto *ldescr = static_cast<CsrDescr *>(L_->GetMatDescr())->sp_descr_;
  auto fill_lower = HIPSPARSE_FILL_MODE_LOWER;
  auto diag_type = HIPSPARSE_DIAG_TYPE_NON_UNIT;

  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(ldescr, HIPSPARSE_SPMAT_FILL_MODE,
                                           &fill_lower, sizeof(fill_lower)));
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(ldescr, HIPSPARSE_SPMAT_DIAG_TYPE,
                                           &diag_type, sizeof(diag_type)));

  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrM));
  CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrM, baseIdx));
  CHECK_CUSPARSE(hipsparseSetMatType(descrM, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CHECK_CUSPARSE(hipsparseSetMatFillMode(descrM, HIPSPARSE_FILL_MODE_LOWER));
  CHECK_CUSPARSE(hipsparseSetMatDiagType(descrM, HIPSPARSE_DIAG_TYPE_NON_UNIT));
  CHECK_CUSPARSE(hipsparseCreateCsric02Info(&infoM));

  CHECK_CUSPARSE(hipsparseDcsric02_bufferSize(cusparseHandle, m, nnz, descrM,
                                             d_L_values, d_A_rows, d_A_columns,
                                             infoM, &bufferSizeIC));
  CHECK_CUDA(hipMalloc(&d_bufferIC, bufferSizeIC));
  CHECK_CUSPARSE(hipsparseDcsric02_analysis(
      cusparseHandle, m, nnz, descrM, d_L_values, d_A_rows, d_A_columns, infoM,
      HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferIC));
  int structural_zero;
  auto status =
      hipsparseXcsric02_zeroPivot(cusparseHandle, infoM, &structural_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
    AX_THROW_RUNTIME_ERROR("A({},{}) is missing", structural_zero,
                           structural_zero);
  }

  // M = L * L^T
  CHECK_CUSPARSE(hipsparseDcsric02(cusparseHandle, m, nnz, descrM, d_L_values,
                                  d_A_rows, d_A_columns, infoM,
                                  HIPSPARSE_SOLVE_POLICY_NO_LEVEL, d_bufferIC));
  // Find numerical zero
  int numerical_zero;
  CHECK_CUSPARSE(
      hipsparseXcsric02_zeroPivot(cusparseHandle, infoM, &numerical_zero));

  CHECK_CUSPARSE(hipsparseDestroyCsric02Info(infoM));
  CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrM));
  CHECK_CUDA(hipFree(d_bufferIC));

  // after that, we perform analysis and allocate external_buffers.
  auto *spsvDescrL =
      std::static_pointer_cast<SpsvDescr>(spsv_descr_)->spsvDescr_;
  auto *spsvDescrLt =
      std::static_pointer_cast<SpsvDescr>(spsv_t_descr_)->spsvDescr_;
  auto *matL = static_cast<CsrDescr *>(L_->GetMatDescr())->sp_descr_;
  auto d_b = std::static_pointer_cast<details::DnVec>(b_descr_);
  auto d_tmp = std::static_pointer_cast<details::DnVec>(temp_descr_);
  auto d_x = std::static_pointer_cast<details::DnVec>(x_descr_);

  // x = L^-T L^-1 b. decompose the problem into two steps:
  //    1. temp <- L^-1 b
  //    2. x = L^-T temp
  size_t bufferSizeL = 0;
  size_t bufferSizeLt = 0;
  // step 1: L^-1 b => temp
  CHECK_CUSPARSE(hipsparseSpSV_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, d_b->vec_,
      d_tmp->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL,
      &bufferSizeL));
  if (external_buffer_l_) {
    CHECK_CUDA(hipFree(external_buffer_l_));
  }
  CHECK_CUDA(hipMalloc(&external_buffer_l_, bufferSizeL));
  CHECK_CUSPARSE(hipsparseSpSV_analysis(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, d_b->vec_,
      d_tmp->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL,
      external_buffer_l_));

  // step 2: L^-T temp => x
  CHECK_CUSPARSE(hipsparseSpSV_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, d_x->vec_,
      d_tmp->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLt,
      &bufferSizeLt));
  if (external_buffer_lt_) {
    CHECK_CUDA(hipFree(external_buffer_lt_));
  }
  CHECK_CUDA(hipMalloc(&external_buffer_lt_, bufferSizeLt));
  CHECK_CUSPARSE(hipsparseSpSV_analysis(
      cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, d_x->vec_,
      d_tmp->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLt,
      external_buffer_lt_));
}

void ImplIcCsrGpu::Solve(ConstRealBufferView b, RealBufferView x) const {
  auto *cusparseHandle = details::get_cusparse_handle();
  int m = static_cast<size_t>(mat_->Rows());
  int nnz = static_cast<size_t>(prod(mat_->Values()->Shape()));

  auto *spsvDescrL =
      std::static_pointer_cast<SpsvDescr>(spsv_descr_)->spsvDescr_;
  auto *spsvDescrLt =
      std::static_pointer_cast<SpsvDescr>(spsv_t_descr_)->spsvDescr_;
  auto *matL = static_cast<CsrDescr *>(L_->GetMatDescr())->sp_descr_;
  auto d_b = std::static_pointer_cast<details::DnVec>(b_descr_);
  auto d_tmp = std::static_pointer_cast<details::DnVec>(temp_descr_);
  auto d_x = std::static_pointer_cast<details::DnVec>(x_descr_);
  auto temp = mid_result_->View();
  mid_result_->SetBytes(0);

  CHECK_CUSPARSE(
      hipsparseDnVecSetValues(d_b->vec_, const_cast<Real *>(b.Data())));
  CHECK_CUSPARSE(hipsparseDnVecSetValues(d_x->vec_, x.Data()));
  CHECK_CUSPARSE(
      hipsparseDnVecSetValues(d_tmp->vec_, const_cast<Real *>(temp.Data())));

  // A \approx L L.T
  // A x = b
  // L L.T x = b
  // 1. L t = b
  // 2. L.T x = t.

  // step 1. solve L^-1 b => temp
  CHECK_CUSPARSE(hipsparseSpSV_solve(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matL, d_b->vec_,
      d_tmp->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

  // step 2. solve L^-T temp => x
  CHECK_CUSPARSE(hipsparseSpSV_solve(
      cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matL, d_tmp->vec_,
      d_x->vec_, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrLt));
}

} // namespace ax::math