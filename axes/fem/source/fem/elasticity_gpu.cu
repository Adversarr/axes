#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/zip_function.h>


#include "ax/fem/elasticity/linear.hpp"
#include "ax/fem/elasticity/neohookean_bw.hpp"
#include "ax/fem/elasticity/stable_neohookean.hpp"
#include "ax/fem/elasticity/stvk.hpp"
#include "ax/fem/elasticity_gpu.cuh"

namespace ax::fem {

template <idx dim> using SvdR = math::decomp::SvdResultImpl<dim, real>;

template <idx dim, template <idx> class ElasticModelTemplate>
struct ElasticityCompute_GPU<dim, ElasticModelTemplate>::Impl {
  thrust::device_vector<idx> seq_;
  thrust::device_vector<math::veci<dim + 1>> elements_;
  thrust::device_vector<math::matr<dim, dim>> deformation_gradient_;
  thrust::device_vector<math::matr<dim, dim>> rinv_gpu_;
  thrust::device_vector<real> rest_volume_gpu_;
  thrust::device_vector<SvdR<dim>> svd_results_;
};

template <idx dim>
__global__ void ComputeDeformationGradient(math::veci<dim + 1> const* elements,
                                           math::vecr<dim> const* pose,
                                           math::matr<dim, dim>* deformation_gradient,
                                           math::matr<dim, dim>* rinv, idx n_elem) {
  idx eid = blockIdx.x * blockDim.x + threadIdx.x;
  if (eid >= n_elem) return;

  math::veci<dim + 1> elem = elements[eid];
  math::vecr<dim> x0 = pose[elem[0]];
  math::matr<dim, dim> Dm;
  for (idx i = 0; i < dim; ++i) {
    Dm.col(i) = pose[elem[i + 1]] - x0;
  }

  deformation_gradient[eid] = Dm * rinv[eid];
}

template <idx dim>
__global__ void ComputeRestPose(math::veci<dim + 1> const* elements, math::vecr<dim> const* pose,
                                math::matr<dim, dim>* rinv, real* rest_volume, idx n_elem) {
  idx eid = blockIdx.x * blockDim.x + threadIdx.x;
  if (eid >= n_elem) return;

  math::veci<dim + 1> elem = elements[eid];
  math::vecr<dim> x0 = pose[elem[0]];
  math::matr<dim, dim> Dm;
#pragma unroll
  for (idx i = 0; i < dim; ++i) {
    Dm.col(i) = pose[elem[i + 1]] - x0;
  }

  rinv[eid] = Dm.inverse();
  real coef = dim == 3 ? 1.0 / 6.0 : 1.0 / 2.0;
  rest_volume[eid] = coef / abs(math::det(rinv[eid]));
}

template <idx dim, template <idx> class ElasticModelTemplate>
ElasticityCompute_GPU<dim, ElasticModelTemplate>::ElasticityCompute_GPU(TriMesh<dim> const& mesh)
    : ElasticityComputeBase<dim>(mesh) {
    hipError_t error = hipSetDevice(0);
    if (error != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(error) << std::endl;
    }
    impl_ = std::make_unique<Impl>();
}

template <idx dim, template <idx> class ElasticModelTemplate>
ElasticityCompute_GPU<dim, ElasticModelTemplate>::~ElasticityCompute_GPU() {
  this->impl_.reset();
}

template <idx dim, template <idx> class ElasticModelTemplate>
bool ElasticityCompute_GPU<dim, ElasticModelTemplate>::UpdateDeformationGradient(
    math::fieldr<dim> const& pose, DeformationGradientUpdate) {
  thrust::host_vector<math::vecr<dim>> pose_cpu(pose.cols());
  for (idx i = 0; i < pose.cols(); ++i) {
    pose_cpu[i] = pose.col(i);
  }
  thrust::device_vector<math::vecr<dim>> pose_gpu(pose_cpu);
  idx n_elem = this->mesh_.GetNumElements();
  ComputeDeformationGradient<dim><<<(n_elem + 127) / 128, 128>>>(
      thrust::raw_pointer_cast(impl_->elements_.data()), thrust::raw_pointer_cast(pose_gpu.data()),
      thrust::raw_pointer_cast(impl_->deformation_gradient_.data()),
      thrust::raw_pointer_cast(impl_->rinv_gpu_.data()), n_elem);
  hipDeviceSynchronize();
  return true;
}

template <idx dim, template <idx> class ElasticModelTemplate>
void ElasticityCompute_GPU<dim, ElasticModelTemplate>::RecomputeRestPose() {
  idx n_elem = this->mesh_.GetNumElements();
  impl_->seq_.resize(n_elem);
  thrust::sequence(impl_->seq_.begin(), impl_->seq_.end());

  // Elements
  impl_->elements_.resize(n_elem);
  thrust::host_vector<math::veci<dim + 1>> elements_host(n_elem);
  for (idx eid = 0; eid < n_elem; ++eid) {
    for (idx i = 0; i < dim + 1; ++i) {
      elements_host[eid][i] = this->mesh_.GetElement(eid)[i];
    }
  }
  thrust::copy(elements_host.begin(), elements_host.end(), impl_->elements_.begin());

  impl_->rest_volume_gpu_.resize(n_elem);
  impl_->rinv_gpu_.resize(n_elem);
  impl_->deformation_gradient_.resize(n_elem);
  impl_->svd_results_.resize(n_elem);

  auto const& pose = this->mesh_.GetVertices();
  thrust::host_vector<math::vecr<dim>> pose_cpu(pose.cols());
  for (idx i = 0; i < pose.cols(); ++i) {
    pose_cpu[i] = pose.col(i);
  }
  thrust::device_vector<math::vecr<dim>> pose_gpu(pose_cpu);
  ComputeRestPose<dim><<<(n_elem + 127) / 128, 128>>>(
      thrust::raw_pointer_cast(impl_->elements_.data()), thrust::raw_pointer_cast(pose_gpu.data()),
      thrust::raw_pointer_cast(impl_->rinv_gpu_.data()),
      thrust::raw_pointer_cast(impl_->rest_volume_gpu_.data()), n_elem);

  auto& rv_cpu= this->rest_volume_;
  auto& rinv_cpu = this->rinv_;
  rv_cpu.resize(1, n_elem);
  rinv_cpu.resize(n_elem);
  thrust::copy(impl_->rest_volume_gpu_.begin(), impl_->rest_volume_gpu_.end(), rv_cpu.data());
  thrust::copy(impl_->rinv_gpu_.begin(), impl_->rinv_gpu_.end(), rinv_cpu.data());
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    AX_LOG(ERROR) << "Error: " << hipGetErrorString(err);
  }
}

template <idx dim, template <idx> class ElasticModelTemplate>
math::field1r ElasticityCompute_GPU<dim, ElasticModelTemplate>::Energy(math::vec2r const& lame) {
  thrust::device_vector<real> energy_device(impl_->deformation_gradient_.size());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.end(), 
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      energy_device.begin(),
      thrust::make_zip_function([lame] __device__(math::matr<dim, dim> const& F,
                                                  real const& rest_volume,
                                                  SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Energy(F, svd) * rest_volume;
      }));

  math::field1r energy(impl_->deformation_gradient_.size());
  thrust::copy(energy_device.begin(), energy_device.end(), energy.data());
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    AX_LOG(ERROR) << "Error: " << hipGetErrorString(err);
  }
  return energy;
};

template <idx dim, template <idx> class ElasticModelTemplate>
math::field1r ElasticityCompute_GPU<dim, ElasticModelTemplate>::Energy(math::field2r const& lame) {
  thrust::device_vector<real> energy_device(impl_->deformation_gradient_.size());
  thrust::device_vector<math::vec2r> lame_device(lame.cols());
  thrust::host_vector<math::vec2r> lame_host(lame.cols());
  for (idx i = 0; i < lame.cols(); ++i) {
    lame_host[i] = lame.col(i);
  }
  thrust::copy(lame_host.begin(), lame_host.end(), lame_device.begin());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.begin(),
                                                   impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.end(),
                                                   impl_->deformation_gradient_.end(), 
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      energy_device.begin(),
      thrust::make_zip_function([] __device__(
          math::vec2r const& lame,
          math::matr<dim, dim> const& F,
          real const& rest_volume,
          SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Energy(F, svd) * rest_volume;
      }));

  math::field1r energy(impl_->deformation_gradient_.size());
  thrust::copy(energy_device.begin(), energy_device.end(), energy.data());
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    AX_LOG(ERROR) << "Error: " << hipGetErrorString(err);
  }
  return energy;
};

template <idx dim, template <idx> class ElasticModelTemplate>
List<elasticity::StressTensor<dim>> ElasticityCompute_GPU<dim, ElasticModelTemplate>::Stress(
    math::vec2r const& lame) {
  thrust::device_vector<elasticity::StressTensor<dim>> stress_device(impl_->deformation_gradient_.size());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.end(),
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      stress_device.begin(),
      thrust::make_zip_function([lame] __device__(math::matr<dim, dim> const& F,
                                                  real const& rest_volume,
                                                  SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Stress(F, svd) * rest_volume;
      }));

  List<elasticity::StressTensor<dim>> stress(impl_->deformation_gradient_.size());
  thrust::copy(stress_device.begin(), stress_device.end(), stress.data());
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    AX_LOG(ERROR) << "Error: " << hipGetErrorString(err);
  }
  return stress;
};

template <idx dim, template <idx> class ElasticModelTemplate>
List<elasticity::StressTensor<dim>> ElasticityCompute_GPU<dim, ElasticModelTemplate>::Stress(
    math::field2r const& lame) {
  thrust::device_vector<elasticity::StressTensor<dim>> stress_device(impl_->deformation_gradient_.size());
  thrust::device_vector<math::vec2r> lame_device(lame.cols());
  thrust::host_vector<math::vec2r> lame_host(lame.cols());
  for (idx i = 0; i < lame.cols(); ++i) {
    lame_host[i] = lame.col(i);
  }
  thrust::copy(lame_host.begin(), lame_host.end(), lame_device.begin());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.begin(),
                                                   impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.end(),
                                                   impl_->deformation_gradient_.end(),
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      stress_device.begin(),
      thrust::make_zip_function([] __device__(
          math::vec2r const& lame,
          math::matr<dim, dim> const& F,
          real const& rest_volume,
          SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Stress(F, svd) * rest_volume;
      }));

  List<elasticity::StressTensor<dim>> stress(impl_->deformation_gradient_.size());
  thrust::copy(stress_device.begin(), stress_device.end(), stress.data());
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    AX_LOG(ERROR) << "Error: " << hipGetErrorString(err);
  }
  return stress;
}

template <idx dim, template <idx> class ElasticModelTemplate>
List<elasticity::HessianTensor<dim>> ElasticityCompute_GPU<dim, ElasticModelTemplate>::Hessian(
    math::field2r const& lame) {
  thrust::device_vector<elasticity::HessianTensor<dim>> hessian_device(impl_->deformation_gradient_.size());
  thrust::device_vector<math::vec2r> lame_device(lame.cols());
  thrust::host_vector<math::vec2r> lame_host(lame.cols());
  for (idx i = 0; i < lame.cols(); ++i) {
    lame_host[i] = lame.col(i);
  }
  thrust::copy(lame_host.begin(), lame_host.end(), lame_device.begin());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.begin(),
                                                   impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(lame_device.end(),
                                                   impl_->deformation_gradient_.end(),
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      hessian_device.begin(),
      thrust::make_zip_function([] __device__(
          math::vec2r const& lame,
          math::matr<dim, dim> const& F,
          real const& rest_volume,
          SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Hessian(F, svd) * rest_volume;
      }));

  List<elasticity::HessianTensor<dim>> hessian(impl_->deformation_gradient_.size());
  thrust::copy(hessian_device.begin(), hessian_device.end(), hessian.data());
  hipDeviceSynchronize();
  return hessian;
}

template <idx dim, template <idx> class ElasticModelTemplate>
List<elasticity::HessianTensor<dim>> ElasticityCompute_GPU<dim, ElasticModelTemplate>::Hessian(
    math::vec2r const& lame) {
  thrust::device_vector<elasticity::HessianTensor<dim>> hessian_device(impl_->deformation_gradient_.size());
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.begin(),
                                                   impl_->rest_volume_gpu_.begin(),
                                                   impl_->svd_results_.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(impl_->deformation_gradient_.end(),
                                                   impl_->rest_volume_gpu_.end(),
                                                   impl_->svd_results_.end())),
      hessian_device.begin(),
      thrust::make_zip_function([lame] __device__(math::matr<dim, dim> const& F,
                                                  real const& rest_volume,
                                                  SvdR<dim> const& svd) {
        return ElasticModel(lame[0], lame[1]).Hessian(F, svd) * rest_volume;
      }));

  List<elasticity::HessianTensor<dim>> hessian(impl_->deformation_gradient_.size());
  thrust::copy(hessian_device.begin(), hessian_device.end(), hessian.data());
  hipDeviceSynchronize();
  return hessian;
}

// NOTE: Currently, ARAP relies on Jacobi SVD.
// template class ElasticityCompute_GPU<2, elasticity::StableNeoHookean>;
// template class ElasticityCompute_GPU<2, elasticity::NeoHookeanBW>;
// template class ElasticityCompute_GPU<2, elasticity::StVK>;
// template class ElasticityCompute_GPU<2, elasticity::Linear>;
// template class ElasticityCompute_GPU<2, elasticity::IsotropicARAP>;

template class ElasticityCompute_GPU<3, elasticity::StableNeoHookean>;
// template class ElasticityCompute_GPU<3, elasticity::NeoHookeanBW>;
// template class ElasticityCompute_GPU<3, elasticity::StVK>;
// template class ElasticityCompute_GPU<3, elasticity::Linear>;
// template class ElasticityCompute_GPU<3, elasticity::IsotropicARAP>;

}  // namespace ax::fem
