#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/zip_function.h>
#include <cstdio>
#include <complex>
template <typename T> auto arg(T x) { return std::arg(x); }

#include "ax/math/common.hpp"
using namespace ax;
using namespace ax::math;

using real = double;

__global__ void GpuAddKernel(const int num, real* x, real* y) {
    const int thread_grid_idx = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    const int num_threads_in_grid = static_cast<int>(blockDim.x * gridDim.x);
    for (int i = thread_grid_idx; i < num; i += num_threads_in_grid) y[i] += x[i];
}

int main() {
  // Test whether cuda is available.
  int count;
  hipError_t err = hipGetDeviceCount(&count);
  if (err == hipErrorNoDevice) {
    printf("No CUDA device found.\n");
  } else if (err == hipErrorInsufficientDriver) {
    printf("CUDA driver is insufficient.\n");
  } else if (err == hipErrorNoDevice) {
    printf("CUDA device is not available.\n");
  } else {
    printf("CUDA is available.\n");
  }

  printf("CUDA device count: %d\n", count);
  int runtime_version;
  hipRuntimeGetVersion(&runtime_version);
  printf("CUDA runtime version: %d\n", runtime_version);

  int driver_version;
  hipDriverGetVersion(&driver_version);
  printf("CUDA driver version: %d\n", driver_version);

  hipDeviceProp_t props;
  cudaGetDeviceProperties_v2(&props, 0);
  printf("Device name: %s\n", props.name);

  real *x, *y;
  err = hipMalloc((void**) & x, 100 * sizeof(real));
  if (err != hipSuccess) {
    printf("Failed to allocate memory for x.\n");
    return 1;
  }
  err = hipMalloc((void**)&y, 100 * sizeof(real));
  if (err != hipSuccess) {
    printf("Failed to allocate memory for y.\n");
    return 1;
  }
  GpuAddKernel<<<1, 32>>>(100, x, y);
  hipDeviceSynchronize();


  thrust::host_vector<vec2r> h(102400);
  for (int i = 0; i < 102400; ++i) h[i] .setRandom();
  thrust::device_vector<vec2r> a = h;
  thrust::device_vector<vec2r> b = h;
  thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(a.begin(), b.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(a.end(), b.end())),
      a.begin(),
      thrust::make_zip_function(thrust::plus<vec2r>()));
  hipDeviceSynchronize();

  constexpr idx dim = 3;

  thrust::device_vector<idx> seq_;
  thrust::device_vector<math::veci<dim + 1>> elements_;
  thrust::device_vector<math::matr<dim, dim>> deformation_gradient_;
  thrust::device_vector<math::matr<dim, dim>> rinv_gpu_;
  thrust::device_vector<real> rest_volume_gpu_;

  seq_.resize(100);
  elements_.resize(100);
  deformation_gradient_.resize(100);
  rinv_gpu_.resize(100);
  rest_volume_gpu_.resize(100);

  thrust::sequence(thrust::device, seq_.begin(), seq_.end());

  printf("Done.\n");
  return 0;
}
